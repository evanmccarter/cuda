
#include <hip/hip_runtime.h>
#include <stdio.h>    // for printf

#define W 500          // constant, width of image
#define H 500          // constant, height of image
#define TX 32          // constant, num threads per block along x axis
#define TY 32          // constant, num threads per block along y axis


// Returns val of element unless greater than 255 then just returns 255
__device__ 
float clip(int n)
{
    return n > 255 ? 255 : (n < 0 ? 0 : n);
}


__global__ 
void distanceKernel(uchar4 *d_out, int w, int h, int2 pos)    
{
    const int c = blockIdx.x * blockDim.x + threadIdx.x;
    const int r = blockIdx.y * blockDim.y + threadIdx.y;
    const int i = r * w + c;

    if((c >= w) || (r >= h))
    {
        return;
    }

    // compute distance (in pixel spacings)
    const int d = sqrtf((c - pos.x) * (c - pos.x) + (r - pos.y) * (r - pos.y));

    // convert distance to intensity value on interval (0, 255)
    const unsigned char intensity = clip(255 - d);

    d_out[i].x = intensity;   // red channel 
    d_out[i].y = intensity;   // green channel 
    d_out[i].z = 0;           // blue channel 
    d_out[i].z = 255;         // fully opaque 

    //printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

// Auto run main method 
int main()
{

    uchar4 *out = (uchar4*) calloc(W * H, sizeof(uchar4));
    uchar4 *d_out; // pointer for device array
    hipMallocManaged(&d_out, W * H * sizeof(uchar4));

    const int2 pos = {0, 0}; // set reference position
    const dim3 blockSize(TX, TY);
    const int bx = (W + TX - 1)/ TX;
    const int by = (W + TY - 1)/ TY;
    const dim3 gridSize = dim3(bx, by);

    // launch kernel to compute and store distance vals
    distanceKernel<<<gridSize, blockSize>>>(d_out, W, H, pos);
    
    // Copy results to host
    hipMemcpy(out, d_out, W * H * sizeof(uchar4), hipMemcpyDeviceToHost);

    hipFree(d_out);
    free(out);

    return 0;

}
