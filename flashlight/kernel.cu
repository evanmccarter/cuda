#include "hip/hip_runtime.h"
#include "kernel.h"

#define TX 32          // constant, num threads per block along x axis
#define TY 32          // constant, num threads per block along y axis


// Returns val of element unless greater than 255 then just returns 255
__device__
unsigned char clip(int n) 
{ 
    return n > 255 ? 255 : (n < 0 ? 0 : n); 
}

__global__
void distanceKernel(uchar4 *d_out, int w, int h, int2 pos) 
{
    const int c = blockIdx.x*blockDim.x + threadIdx.x;
    const int r = blockIdx.y*blockDim.y + threadIdx.y;
    const int i = c + r * w;   // 1D indexing

    if ((c >= w) || (r >= h))
    { 
        return; // Check if within image bounds
    }

    // compute distance (in pixel spacings)
    const int dist = sqrtf((c - pos.x)*(c - pos.x) + 
                           (r - pos.y)*(r - pos.y));

    // convert distance to intensity value on interval (0, 255)
    const unsigned char intensity = clip(255 - dist);

    d_out[i].x = intensity;   // red channel 
    d_out[i].y = intensity;   // green channel 
    d_out[i].z = 0;           // blue channel 
    d_out[i].w = 255;         // fully opaque 
}

void kernelLauncher(uchar4 *d_out, int w, int h, int2 pos) 
{
    const dim3 blockSize(TX, TY);
    const dim3 gridSize = dim3((w + TX - 1)/TX, (h + TY - 1)/TY);
    distanceKernel<<<gridSize, blockSize>>>(d_out, w, h, pos);
}
