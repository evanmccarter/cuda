
#include <hip/hip_runtime.h>
#include <stdio.h>    // for printf
// #include <math.h>  // math lib already included 

#define N 64          // constant, array len
#define TPB 32        // constant, threads per block

// converts int to evenly spaced floats 
// ie) .1, .2, ..., .5, ..., .9
__device__ float scale(int i, int n)
{
    return ((float) i) / (n - 1);
}


// Computes distance between 2 points on a line
__device__ float distance(float x1, float x2)
{
    return sqrt((x2 - x1) * (x2 - x1));
}


__global__ void distanceKernal(float *d_out, float ref, int len)    
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const float x = scale(i, len);
    d_out[i] = distance(x, ref);
    printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

// Auto run main method
int main()
{
    // reference point to be measured from
    const float ref = 0.5f;

    // declare pointeer to array of floats
    float *d_out;

    // allocate device memory tostore output array
    hipMalloc(&d_out, N * sizeof(float));

    // launch kernal to copute and store distance values
    distanceKernal<<<N/TPB, TPB>>>(d_out, ref, N);    

    // free memory
    hipFree(d_out);

    return 0;
}
