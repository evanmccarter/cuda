
#include <hip/hip_runtime.h>
#include <stdio.h>    // for printf

#define N 64          // constant, threads per block
#define TPB 32        // constant, threads per block

// converts int to evenly spaced floats 
// ie) .1, .2, ..., .5, ..., .9
float scale(int i, int n)
{
    return ((float) i) / (n - 1);
}

// Computes distance between 2 points on a line
__device__ 
float distance(float x1, float x2)
{
    return sqrt((x2 - x1) * (x2 - x1));
}


__global__ 
void distanceKernel(float *d_out, float *d_in, float ref)    
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const float x = d_in[i];
    d_out[i] = distance(x, ref);
    printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

// Auto run main method 
int main()
{

    float ref = 0.5f;

    // declare pointers to device arrays
    float *in = 0;
    float *out = 0;

    // allocate device memory to device arrays
    hipMallocManaged(&in, N * sizeof(float));
    hipMallocManaged(&out, N * sizeof(float));

    // launch kernel to copute and store distance values
    for(int i = 0; i < N; i++)
    {
        in[i] = scale(i, N);
    }

    // launch kernel to compute and store distance vals
    distanceKernel<<<N/TPB, TPB>>>(out, in, ref);
    hipDeviceSynchronize();
    
    // free memory for device arrays
    hipFree(in);
    hipFree(out);

    return 0;

}
